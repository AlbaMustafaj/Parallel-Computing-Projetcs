#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>



__global__ void compute ( int* d_arr1 , int* d_arr2, double* dotPRoduct , double* lenghtOfVector1 , double* lenghtOfVector2 , int dataForEach , int blockSize) {

   int index =  blockSize * blockIdx.x + threadIdx.x; 
   //initialize the corresponding vectors' indexes with 0
   dotPRoduct[index] = 0;
   lenghtOfVector1[index] = 0;
   lenghtOfVector2[index] = 0;
  // int dotprod = 0;
   //int length1 = 0;
   //int length2 = 0;
   int i;
   for ( i = 0; i < dataForEach; i++) {
        dotPRoduct[index] += (d_arr1[dataForEach * index + i] * d_arr2[dataForEach * index + i]);
        lenghtOfVector1[index] += (d_arr1[dataForEach * index + i] * d_arr1[dataForEach * index + i]);
        lenghtOfVector2[index] += (d_arr2[dataForEach * index + i] * d_arr2[dataForEach * index + i]);	

   }
 
}


__host__ int* read(int length, FILE* f){
	int i;
	int* arr = (int*) malloc(length * sizeof(int));
    char ch[150];
	for (i = 0; i < length; i++){
    char* element = fgets(ch, sizeof(ch), f);
	        arr[i] = atoi(element);                 
    }
	return arr;
}

int main(int argc, char **argv)
{
    //get array size
	char* arrSize = argv[1];
	char* blockS = argv[2];
	int size = atoi(arrSize);
	int blockSize = atoi (blockS);
	int blockNumber = size / blockSize;
	
	printf ( "Info\n");
    printf( "——————\n");
    printf("Number of elements: %d \n", size);
    printf("Number of threads per block: %d\n", blockSize);
    printf ("Number of blocks will be created: %d\n\n", blockNumber);
	
    //main arrays
    int* arr1;
    int* arr2;
	double* h_dotProduct;
	double* h_lenghtOfVector1;
	double* h_lenghtOfVector2;
		
	
	// helper arrays for computation
	int* d_arr1;
	int* d_arr2;
	double* dotPRoduct;
	double* lenghtOfVector1;
	double* lenghtOfVector2;
	
	
	//cosine of angle
	double cos;

    //helping cosine formula
    long double dotProd = 0;
    long double sq1 = 0;
    long double sq2 = 0;
    double angle;
	//end

    arr1= (int *)malloc(sizeof(int)*size);
    arr2 = (int *)malloc(sizeof(int)*size);
	h_dotProduct = (double *) malloc((blockSize *blockNumber)*sizeof(double));
	h_lenghtOfVector1 = (double *) malloc((blockSize *blockNumber)*sizeof(double));
	h_lenghtOfVector2 = (double *) malloc((blockSize *blockNumber)*sizeof(double));

    //generate arrays
	clock_t arrGenerationStart, arrGenerationEnd;
	arrGenerationStart = clock();
	if (argc == 3) {
	int iter;
    srand(time(NULL));
    for (iter = 0; iter < size; iter++) {
      arr1[iter] = rand()%10000;
      arr2[iter] = rand()%10000;
      //printf( " %d %d \n",iter, arr1[iter]);
   }
    arrGenerationEnd = clock();
  }
   else if(argc == 4){	 //there is a file
		FILE* f = fopen(argv[3],"r");
		char ch[150];
		char* s = fgets(ch, sizeof(ch), f);
		size = atoi(s); //first number is size (already given in first parameter)
		
		arrGenerationStart = clock();
		arr1 = read(size,f);
		arr2 = read(size,f);
		arrGenerationEnd = clock();
	}
	else{
		printf("Something is wrong with your parameters");
		exit(EXIT_SUCCESS);
	}
   
   
   clock_t cpuFuncStart, cpuFuncEnd;
	cpuFuncStart = clock();
	long double dott = 0;
    long double sqr1 = 0;
    long double sqr2 = 0;
    double cosine;
    double anGle;
	int c;
    for (c =0; c < size; c++) {
        dott += (arr1[c] * arr2[c]);
        sqr1 += arr1[c] * arr1[c];
        sqr2 += arr2[c] * arr2[c];

    }
    // printf("dot product is %Lf \n" ,  dott);

    sqr1 = sqrtl(sqr1);
    //printf("sq1 is %Lf \n" , sqr1);
    sqr2 = sqrtl(sqr2);
   // printf("sq2 is %Lf \n" , sqr2);

    dott = dott / (sqr1 * sqr2);
    cosine = (double) dott;
   // printf("cos is %f \n" , cosine);

    anGle = (double) acos(cosine);
    anGle = 180 * anGle / 3.14;

    //printf("angle is %f\n" , anGle);

	cpuFuncEnd = clock();
   
   //allocate memory in device 
   hipMalloc( &d_arr1, size*sizeof(int));
   hipMalloc( &d_arr2, size*sizeof(int));
   hipMalloc( &dotPRoduct,((blockSize *blockNumber)*sizeof(double)));
   hipMalloc( &lenghtOfVector1, ((blockSize *blockNumber)*sizeof(double)));
   hipMalloc( &lenghtOfVector2, ((blockSize *blockNumber)*sizeof(double)));
   
   //copy arrays
   clock_t hostToDeviceTrasfeerStart, hostToDeviceTrasfeerEnd;
   hostToDeviceTrasfeerStart = clock();
   hipMemcpy(d_arr1, arr1, size*sizeof(int), hipMemcpyHostToDevice);
   hipMemcpy(d_arr2, arr2, size*sizeof(int), hipMemcpyHostToDevice);
   hostToDeviceTrasfeerEnd = clock();

   
   //calculate data for each thread
   int dataForEach =  size / (blockNumber * blockSize);
   
   //call  global function
   clock_t kernelStart, kernelEnd;
     kernelStart = clock();
   compute<<<blockNumber,blockSize>>> (d_arr1, d_arr2, dotPRoduct,lenghtOfVector1,lenghtOfVector2, dataForEach, blockSize);
   hipDeviceSynchronize();
   kernelEnd = clock();
   
   clock_t deviceToHostStart, deviceToHostEnd;
   deviceToHostStart = clock();
   hipMemcpy(h_dotProduct,dotPRoduct, (blockSize *blockNumber)*sizeof(double), hipMemcpyDeviceToHost);
   deviceToHostEnd = clock();
   hipMemcpy(h_lenghtOfVector1,lenghtOfVector1, (blockSize *blockNumber)*sizeof(double), hipMemcpyDeviceToHost);
   hipMemcpy(h_lenghtOfVector2,lenghtOfVector2, (blockSize *blockNumber)*sizeof(double), hipMemcpyDeviceToHost);
      
  // collect data   
   int a;
   for( a =0; a < (blockSize *blockNumber); a++) {
 //      // printf( " %f " ,h_lenghtOfVector1[a]);
		
       dotProd += h_dotProduct[a];
		sq1 += h_lenghtOfVector1[a];
		sq2 += h_lenghtOfVector2[a]; 
  }
   
   //if size not divisible by (blockSize * blockNumber)
   int delivered = dataForEach * blockSize * blockNumber;
   int remaining = size - delivered;
   if (remaining > 0) {
     for (int i = delivered; i < size; i++) {
	    dotProd += arr1[i] * arr2[i];
		sq1 +=  arr1[i] * arr1[i];
		sq2 += arr2[i] * arr2[i];
	 }
   }
    sq1 = sqrtl(sq1);
   // printf("sq1 is %Lf \n" , sq1);
    sq2 = sqrtl(sq2);
   // printf("sq2 is %Lf \n" , sq2);

    dotProd = dotProd / (sq1 * sq2);
    cos = (double) dotProd;
    //printf("cos is %f \n" , cos);

    angle = acos(cos);
    angle = 180 * angle / 3.14;

  //  printf("angle is %f \n" , angle);
	
	printf("Time\n");
    printf("——————\n");
    printf("Time for the array generation : %f ms\n", ((double) (arrGenerationEnd - arrGenerationStart)) /CLOCKS_PER_SEC * 1000);
    printf("Time for the CPU function : %f ms\n", ((double) (cpuFuncEnd - cpuFuncStart)) /CLOCKS_PER_SEC * 1000);
    printf("Time for the Host to Device transfer : %f ms\n", ((double) (hostToDeviceTrasfeerEnd - hostToDeviceTrasfeerStart)) /CLOCKS_PER_SEC * 1000);
    printf("Time for the kernel execution : %f ms\n", ((double) (kernelEnd - kernelStart)) /CLOCKS_PER_SEC * 1000);
    printf("Time for the Device to Host transfer : %f ms\n" , ((double) (deviceToHostEnd - deviceToHostStart)) /CLOCKS_PER_SEC *1000);
    double gpuTime = (hostToDeviceTrasfeerEnd - hostToDeviceTrasfeerStart) + (kernelEnd - kernelStart) + (deviceToHostEnd - deviceToHostStart);
	printf("Total execution time for GPU : %f ms\n\n ", ((double) gpuTime / CLOCKS_PER_SEC) *1000);
	printf("Results\n");
	printf("——————\n");
	printf("CPU result: %f \n", anGle);
	printf("GPU result: %f \n", angle);
    return 0;
	
}